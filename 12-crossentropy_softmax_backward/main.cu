#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <corecrt_math.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <chrono>
#include "../00-common/common.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <vcruntime.h>
#include <corecrt_math_defines.h>

// turn on bf16 as default, done up here for now
#define ENABLE_BF16

#if defined(ENABLE_BF16)
typedef __hip_bfloat16 floatX;
typedef __hip_bfloat16 floatN;
#elif defined(ENABLE_FP16)
typedef half floatX;
typedef half floatN;
#else
typedef float floatX;
typedef float floatN;
#endif

typedef Packed128<floatX> x128;

#define CHECK(call)                                   \
do                                                    \
{                                                     \
    const hipError_t error_code = call;              \
    if (error_code != hipSuccess)                    \
    {                                                 \
        printf("CUDA Error:\n");                      \
        printf("    File:       %s\n", __FILE__);     \
        printf("    Line:       %d\n", __LINE__);     \
        printf("    Error code: %d\n", error_code);   \
        printf("    Error text: %s\n",                \
            hipGetErrorString(error_code));          \
        exit(1);                                      \
    }                                                 \
} while (0)

// CPU code reference
void crossentropy_softmax_backward_cpu(float* dlogits,
                                       const float* dlosses,
                                       const float* probs,
                                       const int* targets,
                                       int B,
                                       int T,
                                       int C)
{
    for (int b = 0; b < B; b++)
    {
        for (int t = 0; t < T; t++)
        {
            float* dlogits_bt = dlogits + b * T * C + t * C;
            const float* probs_bt = probs + b * T * C + t * C;
            float dloss = dlosses[b * T + t];
            int ix = targets[b * T + t];
            for (int i = 0; i < C; i++)
            {
                float p = probs_bt[i];
                float indicator = i == ix ? 1.0f : 0.0f;
                dlogits_bt[i] += (p - indicator) * dloss;
            }
        }
    }
}

__global__ void crossentropy_softmax_backward_kernel1(float* dlogits,
                                                      const float* dlosses,
                                                      const float* probs,
                                                      const int* targets,
                                                      int B,
                                                      int T,
                                                      int C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < B * T * C)
    {
        int b = i / (T * C);
        int t = (i / C) % T;
        int c = i % C;
        float* dlogits_bt = dlogits + b * T * C + t * C;
        const float* probs_bt = probs + b * T * C + t * C;
        float dloss = dlosses[b * T + t];
        int ix = targets[b * T + t];
        float p = probs_bt[c];
        float indicator = c == ix ? 1.0f : 0.0f;
        dlogits_bt[c] += (p - indicator) * dloss;
    }

}

void crossentropy_softmax_backward1(float* dlogits,
                                    const float* dlosses,
                                    const float* probs,
                                    const int* targets,
                                    int B,
                                    int T,
                                    int C,
                                    const int block_size)
{
    const int grid_size = ceil_div(B * T * C, block_size);
    crossentropy_softmax_backward_kernel1<<<grid_size, block_size>>>(dlogits, dlosses, probs, targets, B, T, C);
    cudaCheck(hipGetLastError());
}

void crossentropy_softmax_backward(int kernel_num,
                                   float* dlogits,
                                   const float* dlosses,
                                   const float* probs,
                                   const int* targets,
                                   int B,
                                   int T,
                                   int C,
                                   const int block_size)
{
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventRecord(start));
    switch (kernel_num)
    {
        case 1:
            crossentropy_softmax_backward1(dlogits, dlosses, probs, targets, B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
    cudaCheck(hipGetLastError());
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("[kernel%d]Checking block size = %d,elapsed_time = %g ms.\n", kernel_num, block_size, elapsed_time);
}

int main(int argc, char **argv)
{
    srand(0);

    int B = 8;
    int T = 1024;
    int V = 50257;

    float* probs = make_random_float(B * T * V);
    int* targets = make_random_int(B * T, V);
    float* dlosses = make_random_float(B * T);
    float* dlogits = make_zeros_float(B * T * V);

    float* d_probs;
    int* d_targets;
    float* d_dlosses;
    float* d_dlogits;

    cudaCheck(hipMalloc(&d_probs, B * T * V * sizeof(float)));
    cudaCheck(hipMalloc(&d_targets, B * T * sizeof(int)));
    cudaCheck(hipMalloc(&d_dlosses, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_dlogits, B * T * V * sizeof(float)));
    cudaCheck(hipMemcpy(d_probs, probs, B * T * V * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_dlosses, dlosses, B * T * sizeof(float), hipMemcpyHostToDevice));

    // cpu
    crossentropy_softmax_backward_cpu(dlogits, dlosses, probs, targets, B, T, V);

    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int i = 0; i < sizeof(block_sizes) / sizeof(int); i++)
    {
        cudaCheck(hipMemset(d_dlogits, 0, B * T * V * sizeof(float)));
        crossentropy_softmax_backward(1, d_dlogits, d_dlosses, d_probs, d_targets, B, T, V, block_sizes[i]);
        float tol = 1e-5;
        validate_result(d_dlogits, dlogits, "d_dlogits", B * T * V, tol);
    }

    // free memory
    free(probs);
    free(targets);
    free(dlosses);
    free(dlogits);
    cudaCheck(hipFree(d_probs));
    cudaCheck(hipFree(d_targets));
    cudaCheck(hipFree(d_dlosses));
    cudaCheck(hipFree(d_dlogits));

    return 0;
}

